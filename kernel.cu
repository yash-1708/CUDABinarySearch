#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#define P 128

//main kernel that runs on GPU
__global__ void BinSearch(int *mainVec, int *toSearch, bool *isPresent, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int key = toSearch[i];
	bool flag = false;
	int min = 0;
	int max = N;
	int mid = (min + max) / 2;
	while (min != mid) {
		if (mainVec[mid] == key) {
			flag = true;
			break;
		}
		else if (key < mainVec[mid]){
			max = mid;
			min = min;
		}
		else {
			min = mid;
			max = max;
		}
		mid = (min + max) / 2;
	}
	isPresent[i] = flag;
}

bool serial_BinSearch(int* mainVec, int* toSearch, int i, int N) {
	int key = toSearch[i];
	bool flag = false;
	int min = 0;
	int max = N;
	int mid = (min + max) / 2;
	while (min != mid) {
		if (mainVec[mid] == key) {
			flag = true;
			break;
		}
		else if (key < mainVec[mid]) {
			max = mid;
			min = min;
		}
		else {
			min = mid;
			max = max;
		}
		mid = (min + max) / 2;
	}
	return flag;
}


//random vector generator
void randomInts(int* a, int N) {
	for (int i = 0; i < N; i++){
		a[i] = rand() % 100;
	}
}

int main() {
	clock_t start, stop;

	int N = 100;//number of elements in main vector
	int M = 123400;//number of elements to be searched
	
	int* mainVec = (int*)malloc(N * sizeof(int));
	int* toSearch = (int*)malloc(M * sizeof(int));
	bool* isPresent = (bool*)malloc(M * sizeof(bool));
	bool* serial_isPresent = (bool*)malloc(M * sizeof(bool));

	//randomInts(mainVec, N);
	for (size_t i = 0; i < N; i++){
		mainVec[i] = i;
	}
	randomInts(toSearch, M);

	/*printf("\n\nMain Vector : \n");
	for (size_t i = 0; i < N; i++)
	{
		printf("%d \n", mainVec[i]);
	}
	
	printf("\n\nTo Search Vector : \n");
	for (size_t i = 0; i < M; i++)
	{
		printf("%d \n", toSearch[i]);
	}*/

	int* d_mainVec;
	int* d_toSearch;
	bool* d_isPresent;

	hipMalloc(&d_mainVec, N * sizeof(int));
	hipMalloc(&d_toSearch, M * sizeof(int));
	hipMalloc(&d_isPresent, M * sizeof(bool));

	hipMemcpy(d_mainVec, mainVec, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_toSearch, toSearch, M * sizeof(int), hipMemcpyHostToDevice);

	start = std::clock();
	BinSearch<<<(P+M-1)/P,M>>>(d_mainVec, d_toSearch, d_isPresent, N);
	hipDeviceSynchronize();
	stop = std::clock();
	long float timeP = stop - start;

	hipMemcpy(isPresent, d_isPresent, M * sizeof(bool), hipMemcpyDeviceToHost);

	printf("\n\n\n%3.3f", timeP);
	/*printf("\n************************PARALLEL*****************************\n");
	for (size_t i = 0; i < M; i++){
		if (isPresent[i] == true) {
			printf("\n isPresent[%d] : true", i);
		}
		else {
			printf("\n isPresent[%d] : false", i);
		}
	}
	printf("\n*************************************************************\n");*/

	start = std::clock();
	for (size_t i = 0; i < M; i++){
		serial_isPresent[i] = serial_BinSearch(mainVec,toSearch,i,N);
	}
	stop = std::clock();
	long float timeN = stop - start;
	printf("\n\n\n%3.3f", timeN);
	/*
	printf("\n*****************************SERIES**************************\n");
	for (size_t i = 0; i < M; i++) {
		if (serial_isPresent[i] == true) {
			printf("\n isPresent[%d] : true", i);
		}
		else {
			printf("\n isPresent[%d] : false", i);
		}
	}
	printf("\n*************************************************************\n");*/

	//getting GPU properties and storing in prop
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int cores = prop.multiProcessorCount * 128;
	float totalCost = cores * timeP;

	//results printing
	printf("\n***********************************************************************************************************************\n");
	printf("M \t\t\t Nor Time \t Par Time \t Cores \t\t Tot Cost \t Speedup \t Efficiency \n");
	printf("%-20d \t %-7.3f \t %-7.3f \t %-10d \t %-7.3f \t %-7.3f \t %-5.5f \n", M, timeN, timeP, cores, totalCost, timeN / timeP, timeN / (timeP * cores));
	printf("\n***********************************************************************************************************************\n");
	
	//free memory
	hipFree(d_mainVec);
	hipFree(d_toSearch);
	hipFree(d_isPresent);
	free(mainVec);
	free(toSearch);
	free(isPresent);
	free(serial_isPresent);

	return  0;
}